#include "hip/hip_runtime.h"
#include "util.h"
#include "handler.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <math.h>

__device__ __forceinline__ int get_index(int X, int Y, int Z, int C, int x, int y, int z) {
  return z * (C * X * Y) + y * X + x;
}

__device__ __forceinline__ void add_c(float const *in, float *out, int slicesizein, int slicesizeout, int C) {
  // *out = *in + *out;
	for (size_t c(0); c < C; ++c)
		out[c * slicesizeout] += in[c * slicesizein];
}

__device__ __forceinline__ void copy_c(float const *in, float *out, int slicesizein, int slicesizeout, int C) {
  // *out = *in;
  for (size_t c(0); c < C; ++c)
   out[c * slicesizeout] = in[c * slicesizein];
}

__global__ void normal_kernel(int seed, float *data, int n, float mean, float std) {
  if (threadIdx.x != 0) return;
  hiprandState state;

  hiprand_init(seed, 0, 0, &state);
  for (size_t i(0); i < n; ++i)
    data[i] = hiprand_normal(&state) * std + mean;
}

__global__ void normal_kerneld(int seed, double *data, int n, double mean, double std) {
  if (threadIdx.x != 0) return;
  hiprandState state;
  hiprand_init(seed, 0, 0, &state);
  for (size_t i(0); i < n; ++i)
    data[i] = hiprand_normal_double(&state) * std + mean;
}

__global__ void add_normal_kernel(int seed, float *data, int n, float mean, float std) {
  if (threadIdx.x != 0) return;
  hiprandState state;

  hiprand_init(seed, 0, 0, &state);
  for (size_t i(0); i < n; ++i)
    data[i] += hiprand_normal(&state) * std + mean;
}

__global__ void add_normal_kerneld(int seed, double *data, int n, double mean, double std) {
  if (threadIdx.x != 0) return;
  hiprandState state;
  hiprand_init(seed, 0, 0, &state);
  for (size_t i(0); i < n; ++i)
    data[i] += hiprand_normal_double(&state) * std + mean;
}

template <>
void init_normal<float>(float *a, int N, float mean, float std) {
     normal_kernel<<<1, 32>>>(rand(), a, N, mean, std);
}

template <>
void init_normal<double>(double *a, int N, double mean, double std) {
     normal_kerneld<<<1, 32>>>(rand(), a, N, mean, std);
}

template <>
void add_normal<float>(float *a, int N, float mean, float std) {
     add_normal_kernel<<<1, 32>>>(rand(), a, N, mean, std);
}

template <>
void add_normal<double>(double *a, int N, double mean, double std) {
     add_normal_kerneld<<<1, 32>>>(rand(), a, N, mean, std);
}

__global__ void rand_init_kernel(int seed, hiprandStatePhilox4_32_10_t *states, int n) {
  int x(threadIdx.x + blockDim.x * blockIdx.x);

  if (x < n)
    hiprand_init(seed, x, 0, &states[x]);
}

__global__ void rand_zero_kernel(float *data, int n, float p, hiprandStatePhilox4_32_10_t *states) {
  int x(threadIdx.x + blockDim.x * blockIdx.x);

  hiprandStatePhilox4_32_10_t &state(states[x]);

  x *= 4;
  float4 vals = hiprand_uniform4(&state);
  for (int i(0); i < 4; ++i, ++x) {
    if (x >= n) return;
    if (reinterpret_cast<float*>(&vals)[i] < p)
      data[x] = 0;
  }
}

void rand_zero(float *data, int n, float p) {
  // assert(n > 1);
  static int n_rand_states = 0;
  static hiprandStatePhilox4_32_10_t* rand_states = 0;

  int const BLOCKSIZE(1024);
  int n_threads = (n + 4 - 1) / 4;

  dim3 dimBlock( BLOCKSIZE );
  dim3 dimGrid( (n_threads + BLOCKSIZE - 1) / BLOCKSIZE );

  if (n_threads > n_rand_states) {
	  if (rand_states) hipFree(rand_states);
	  handle_error(hipMalloc(&rand_states, sizeof(hiprandStatePhilox4_32_10_t) * n_threads));

    rand_init_kernel<<<dimGrid, dimBlock>>>(rand(), rand_states, n_threads);
    n_rand_states = n_threads;
  }


  rand_zero_kernel<<<dimGrid, dimBlock>>>(data, n, p, rand_states);
  handle_error( hipGetLastError() );
  handle_error( hipDeviceSynchronize());
}

__global__ void shift_kernel(float const *in, float *out, int X, int Y, int C, int dx, int dy, float const beta) {
	int x(threadIdx.x + blockDim.x * blockIdx.x);
	int y(x / X);
  x = x % X;

	int x_to(x + dx);
	int y_to(y + dy);
  // int x_to(x);
  // int y_to(y);
  // y = 0;
  // y_to = 0;


	if (x >= X || y >= Y || x_to >= X || y_to >= Y || x_to < 0 || y_to < 0)
		return;
  if (beta>0)
    add_c(in + get_index(X, Y, 1, C, x, y, 0), out + get_index(X, Y, 1, C, x_to, y_to, 0), X * Y, X * Y, C);
  else
    copy_c(in + get_index(X, Y, 1, C, x, y, 0), out + get_index(X, Y, 1, C, x_to, y_to, 0), X * Y, X * Y, C);

}

__global__ void unshift_kernel(float const *in, float *out, int X, int Y, int C, int dx, int dy, float const beta) {
  int x(threadIdx.x + blockDim.x * blockIdx.x);
  int y(x / X);
  x = x % X;

  int x_to(x + dx);
  int y_to(y + dy);

  if (x >= X || y >= Y || x_to >= X || y_to >= Y || x_to < 0 || y_to < 0)
    return;

  if (beta>0)
  	add_c(in + get_index(X, Y, 1, C, x_to, y_to, 0), out + get_index(X, Y, 1, C, x, y, 0), X * Y, X * Y, C);
  else
    copy_c(in + get_index(X, Y, 1, C, x_to, y_to, 0), out + get_index(X, Y, 1, C, x, y, 0), X * Y, X * Y, C);
}

void shift(float const *in, float *out, int X, int Y, int C, int dx, int dy, float const beta) {
	int s = X * Y;// * C;
	int const BLOCKSIZE(1024);

	int dimBlock( BLOCKSIZE );
	int dimGrid( (s  + BLOCKSIZE - 1) / BLOCKSIZE);

	shift_kernel<<<dimGrid, dimBlock>>>(in, out, X, Y, C, dx, dy, beta);
}

void unshift(float const *in, float *out, int X, int Y, int C, int dx, int dy, float const beta) {
	int s = X * Y;// * C;
	int const BLOCKSIZE(1024);

	int dimBlock( BLOCKSIZE );
	int dimGrid( (s  + BLOCKSIZE - 1) / BLOCKSIZE);

	unshift_kernel<<<dimGrid, dimBlock>>>(in, out, X, Y, C, dx, dy, beta);
}
